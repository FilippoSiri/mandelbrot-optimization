
#include <hip/hip_runtime.h>
#include <chrono>
#include <complex.h>
#include <fstream>
#include <iostream>

#ifdef DOUBLE
typedef double __ftype;
#else
typedef float __ftype;
#endif

#ifndef THREADS_X
#define THREADS_X 16
#endif

#ifndef THREADS_Y
#define THREADS_Y 16
#endif

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#ifndef RESOLUTION
#define RESOLUTION 1000
#endif

#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)

#define STEP ((__ftype)RATIO_X / WIDTH)

#ifndef ITERATIONS
#define ITERATIONS 1000 // Maximum number of iterations
#endif

using namespace std;

__global__ void mandelbrot(int *const image) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col >= WIDTH || row >= HEIGHT) {
    return;
  }

  int pos = row * WIDTH + col;

  image[pos] = 0;

  __ftype c_re = col * STEP + MIN_X;
  __ftype c_im = row * STEP + MIN_Y;

  __ftype z_re = 0.0;
  __ftype z_im = 0.0;

  for (int i = 1; i <= ITERATIONS; i++) {
    // xy	=	(a+ib)(c+id)
    // 	    =	(ac-bd)+i(ad+bc).
    // a == c, b == d
    // ==> x * x = (a * a - b * b) + i (2 * a * b)
    __ftype z2_re = z_re * z_re - z_im * z_im;
    __ftype z2_im = 2.0 * z_re * z_im;

    // z = pow(z, 2) + c;
    z_re = z2_re + c_re;
    z_im = z2_im + c_im;

    // |z|2 = x2 + y2.
    __ftype abs2 = z_re * z_re + z_im * z_im;

    // If it is convergent
    if (abs2 >= 4) {
      image[pos] = i;
      return;
    }
  }
}

void handle_error(hipError_t err) {
  if (err != hipSuccess) {
    cerr << "GPUassert: " << hipGetErrorString(err) << endl;
    exit(err);
  }
}

int main(int argc, char **argv) {
  int *const image = new int[HEIGHT * WIDTH];
  const int size = HEIGHT * WIDTH * sizeof(int);

  int *image_gpu = nullptr;

  const auto start = chrono::steady_clock::now();

  handle_error(hipMalloc((void **)&image_gpu, size));

  dim3 threadsPerBlock(THREADS_X, THREADS_Y);
  dim3 numBlocks((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

  mandelbrot<<<numBlocks, threadsPerBlock>>>(image_gpu);

  handle_error(hipMemcpy(image, image_gpu, size, hipMemcpyDeviceToHost));

  const auto end = chrono::steady_clock::now();
  cout << "Time elapsed: "
       << chrono::duration_cast<chrono::milliseconds>(end - start).count()
       << " ms." << endl;

  // Write the result to a file
  ofstream matrix_out;

  if (argc < 2) {
    cout << "Please specify the output file as a parameter." << endl;
    return -1;
  }

  matrix_out.open(argv[1], ios::trunc);
  if (!matrix_out.is_open()) {
    cout << "Unable to open file." << endl;
    return -2;
  }

  for (int row = 0; row < HEIGHT; row++) {
    for (int col = 0; col < WIDTH; col++) {
      matrix_out << image[row * WIDTH + col];

      if (col < WIDTH - 1)
        matrix_out << ',';
    }
    if (row < HEIGHT - 1)
      matrix_out << endl;
  }
  matrix_out.close();

  handle_error(hipFree(image_gpu));
  delete[] image; // It's here for coding style, but useless
  return 0;
}
